#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void kernels::compute_checksums(float* matrix, int rows, int cols, bool checksum_compute_mode)
{
    extern __shared__ float shared_data[]; // shared memory for intermediate sums, as big as the blockdim vector

    int index_orthogonal = checksum_compute_mode == CHECKSUM_COMPUTE_COL ? blockIdx.x : blockIdx.y; // 1 block per row/col checksum, the ortho direction is just the block index in that direction

    float sum = 0.0f;

    // info along the reduction direction
    int index_reduction = checksum_compute_mode == CHECKSUM_COMPUTE_COL ? threadIdx.y : threadIdx.x;
    int blockDim_reduction = checksum_compute_mode == CHECKSUM_COMPUTE_COL ? blockDim.y : blockDim.x;
    int limit_reduction = checksum_compute_mode == CHECKSUM_COMPUTE_COL ? rows : cols;

    // this thread accumulates values in blockdim offsets along the reduction direction
    for (int i = index_reduction; i < limit_reduction; i += blockDim_reduction)
    {
        sum += checksum_compute_mode == CHECKSUM_COMPUTE_COL ? matrix[i * cols + index_orthogonal] : matrix[index_orthogonal * cols + i];
    }

    // this thread stores his partial result into shared memory at his relative offset
    shared_data[index_reduction] = sum;
    __syncthreads(); // other threads do the same for their relative offset, we wait for all threads to finish

    // this threads performs reduction over the shared memory vector
    for (int stride = blockDim_reduction / 2; stride > 0; stride /= 2)
    {
        if (index_reduction < stride && index_reduction + stride < limit_reduction)
        {
            shared_data[index_reduction] += shared_data[index_reduction + stride];
        }
        __syncthreads(); // other threads do the same
    }

    // finally, the first thread writes final reduction result to the matrix directly
    if (index_reduction == 0)
    {
        if (checksum_compute_mode == CHECKSUM_COMPUTE_COL)
        {
            matrix[rows * cols + index_orthogonal] = shared_data[0]; // last row for column checksums
        }
        else
        {
            matrix[index_orthogonal * (cols + 1) + cols] = shared_data[0]; // last column for row checksums
        }
    }
}
