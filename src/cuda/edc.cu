#include "hip/hip_runtime.h"
#include "edc.cuh"
#include "kernels.cuh"
#include "matrix.h"
#include <iomanip>
#include <iostream>
#include <vector>

namespace cuda
{
    EDCResult errors_detect_correct(const float* d_ec_matrix, int rows, int cols, float* d_cc_control, float* d_rc_control, hipStream_t mainStream, hipStream_t secondaryStream, bool* recompute_vertical_checksums, bool* recompute_horizontal_checksums)
    {
        EDCResult edc_res;

        // allocate mismatches index buffers

        int error_xs[EDC_MAX_ERRORS], error_ys[EDC_MAX_ERRORS];
        int *d_error_xs, *d_error_ys;
        hipMalloc(&d_error_xs, EDC_MAX_ERRORS * sizeof(int));
        hipMalloc(&d_error_ys, EDC_MAX_ERRORS * sizeof(int));
        hipMemset(d_error_xs, 0, EDC_MAX_ERRORS * sizeof(int));
        hipMemset(d_error_ys, 0, EDC_MAX_ERRORS * sizeof(int));

        // mismatch info array to avoid multiple allocs and copies
        int* mismatch_info;
        hipHostMalloc(&mismatch_info, 4 * sizeof(int));
        int* d_mismatch_info;
        hipMalloc(&d_mismatch_info, 4 * sizeof(int)); // mismatch_count_x/y, error_x/y
        hipMemset(d_mismatch_info, 0, 4 * sizeof(int));

        CUDA_CHECK

#define MISMATCH_COUNT_X 0
#define ERROR_X 1
#define MISMATCH_COUNT_Y 2
#define ERROR_Y 3

        // depth-first issuing to avoid consecutive kernel scheduling blocking kernel0 signal to copy queue

        kernels::find_checksum_mismatches<<<CEIL_DIV(cols + 1, tileDim.y), tileDim.y, 0, mainStream>>>(d_ec_matrix, rows, cols, d_cc_control, ChecksumsToCompare::COL, &d_mismatch_info[MISMATCH_COUNT_X], d_error_xs, &d_mismatch_info[ERROR_X]);

        hipMemcpyAsync(mismatch_info, d_mismatch_info, 2 * sizeof(float), hipMemcpyDeviceToHost, mainStream);

        kernels::find_checksum_mismatches<<<CEIL_DIV(rows + 1, tileDim.x), tileDim.x, 0, secondaryStream>>>(d_ec_matrix, rows, cols, d_rc_control, ChecksumsToCompare::ROW, &d_mismatch_info[MISMATCH_COUNT_Y], d_error_ys, &d_mismatch_info[ERROR_Y]);

        hipMemcpyAsync(mismatch_info + 2, d_mismatch_info + 2, 2 * sizeof(float), hipMemcpyDeviceToHost, secondaryStream);

        hipStreamSynchronize(mainStream);
        hipStreamSynchronize(secondaryStream);
        CUDA_CHECK

#define AXIS_X ReductionDirection::ALONG_COL
#define AXIS_Y ReductionDirection::ALONG_ROW

        ReductionDirection collinear_axis = mismatch_info[MISMATCH_COUNT_X] <= 1 ? AXIS_Y : AXIS_X; // only 1 (or 0 in case of collinear checksum corruption) mismatch found in x implies the collinear axis must be y and viceversa
        int num_errors = mismatch_info[MISMATCH_COUNT_X] <= 1 ? mismatch_info[MISMATCH_COUNT_Y] : mismatch_info[MISMATCH_COUNT_X];
        int non_discarded = 0;

        if ((mismatch_info[MISMATCH_COUNT_X] | mismatch_info[MISMATCH_COUNT_Y]) == 0)
        {
            // no errors
            edc_res = NO_ERROR;
            goto cleanup;
        }

        if (mismatch_info[ERROR_Y] || mismatch_info[ERROR_X])
        {
            // kernel error (more errors than max allowed)
            edc_res = UNCORRECTABLE_ERROR;
            if (globals::debugPrint)
            {
                COUT << "Kernel error flag raised: (x " << mismatch_info[ERROR_Y] << ", y " << mismatch_info[ERROR_X] << ") mismatches found (max allowed per axis" << EDC_MAX_ERRORS << ")" << ENDL;
            }
            goto cleanup;
        }

        if ((mismatch_info[MISMATCH_COUNT_X] >> 1 & mismatch_info[MISMATCH_COUNT_Y] >> 1) != 0) // one of these must be 1 (shifted right becomes 0) -> collinear axis exists
        {
            // non collinear errors, can't correct
            edc_res = UNCORRECTABLE_ERROR;
            goto cleanup;
        }

        // all errors collinear, or single error: can correct on axis
        edc_res = CORRECTED_ERROR;

        // overwrite d_ec_matrix with corrected vals

        // copy mismatch coords to host
        hipMemcpyAsync(error_xs, d_error_xs, num_errors * sizeof(float), hipMemcpyDeviceToHost, mainStream);
        hipMemcpyAsync(error_ys, d_error_ys, num_errors * sizeof(float), hipMemcpyDeviceToHost, mainStream);

        // allocate host correction checksums
        float* correction_checksums;
        hipHostMalloc(&correction_checksums, num_errors * sizeof(float));

        // allocate host control checksums
        float* control_checksums;
        hipHostMalloc(&control_checksums, num_errors * sizeof(float));

        // allocate host error values
        float* error_values;
        hipHostMalloc(&error_values, num_errors * sizeof(float));

        // allocate host corrected values
        float* corrected_vals;
        hipHostMalloc(&corrected_vals, num_errors * sizeof(float));

        hipStreamSynchronize(mainStream);
        CUDA_CHECK

        for (int i = 0; i < num_errors; ++i)
        {
            // correct collinear coords (one kernel found only 1 mismatch, need to duplicate the single coord)
            if (collinear_axis == AXIS_X) // only 1 y
                error_ys[i] = error_ys[0];
            else // only 1 x
                error_xs[i] = error_xs[0];

            // discard errors on checksum vectors
            if (error_xs[i] == cols)
            {
                *recompute_horizontal_checksums = true;
                continue;
            }

            if (error_ys[i] == rows)
            {
                *recompute_vertical_checksums = true;
                continue;
            }

            non_discarded++;

            // calculate correction

            // copy correction checksum
            hipMemcpyAsync(correction_checksums + i, (collinear_axis == AXIS_X ? d_ec_matrix + rows * (cols + 1) + error_xs[i] : d_ec_matrix + error_ys[i] * (cols + 1) + cols), sizeof(float), hipMemcpyDeviceToHost, mainStream);
            // copy control checksum
            hipMemcpyAsync(control_checksums + i, (collinear_axis == AXIS_X ? d_cc_control + error_xs[i] : d_rc_control + error_ys[i]), sizeof(float), hipMemcpyDeviceToHost, mainStream);
            // copy error value
            hipMemcpyAsync(error_values + i, (void*)(d_ec_matrix + error_ys[i] * (cols + 1) + error_xs[i]), sizeof(float), hipMemcpyDeviceToHost, mainStream);

            hipStreamSynchronize(mainStream);
            CUDA_CHECK

            corrected_vals[i] = correction_checksums[i] - control_checksums[i] + error_values[i];

            // write correction
            hipMemcpyAsync((void*)(d_ec_matrix + error_ys[i] * (cols + 1) + error_xs[i]), corrected_vals + i, sizeof(float), hipMemcpyHostToDevice, mainStream);

            if (globals::debugPrint)
            {
                printf("Found correctable error @ C(%d, %d):\n", error_ys[i] + 1, error_xs[i] + 1); // math notation (row, col)
                COUT << "  mul " << (collinear_axis == AXIS_X ? "col ↓" : "row →") << " checksum = " << FMT_FLOAT(correction_checksums[i]) << ENDL;
                COUT << "  corrected value = " << corrected_vals[i] << ENDL;
            }
        }

        hipStreamSynchronize(mainStream);

        CUDA_CHECK

        hipHostFree(corrected_vals);
        hipHostFree(correction_checksums);
        hipHostFree(error_values);
        hipHostFree(control_checksums);

        CUDA_CHECK

        if (non_discarded == 0)
            edc_res = NO_ERROR;

    cleanup:

        hipHostFree(mismatch_info);
        hipFree(d_mismatch_info);
        hipFree(d_error_xs);
        hipFree(d_error_ys);

        CUDA_CHECK

        return edc_res;
    }
}
