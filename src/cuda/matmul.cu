#include "cuda.cuh"
#include "inferred_matrix_sizes.h"
#include "kernels.cuh"
#include "memsize_string.h"
#include "timer.h"

namespace cuda
{
    void matmul(float* A, float* B, float* C, int rows_A, int cols_A, int cols_B)
    {
        Info info = getInfo();

        float *dA, *dB, *dC;

        hipMalloc(&dA, SIZE_A_BYTES);
        hipMalloc(&dB, SIZE_B_BYTES);
        hipMalloc(&dC, SIZE_C_BYTES);

        {
            ScopedTimer timer("A,B to device", POST);
            hipMemcpy(dA, A, SIZE_A_BYTES, hipMemcpyHostToDevice);
            hipMemcpy(dB, B, SIZE_B_BYTES, hipMemcpyHostToDevice);
        }

        {
            ScopedTimer timer("matmul kernel", POST);

            dim3 tiles(CEIL_DIV(COLS_C, tileDim.x), CEIL_DIV(ROWS_C, tileDim.y));

            dim3 gridDim = tiles;
            dim3 blockDim = tileDim;
            int sharedMemSize = 2 * dim2ToBytes(tileDim);

            kernels::tiled_matmul<<<gridDim, blockDim, sharedMemSize>>>(dA, dB, dC, rows_A, cols_A, cols_B);

            hipDeviceSynchronize();
            CUDA_CHECK
        }

        {
            ScopedTimer timer("C to host", POST);
            hipMemcpy(C, dC, SIZE_C_BYTES, hipMemcpyDeviceToHost);
            CUDA_CHECK
        }

        hipFree(dA);
        hipFree(dB);
        hipFree(dC);
    }
}
