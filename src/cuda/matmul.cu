#include "cuda.cuh"
#include "formatting.h"
#include "inferred_matrix_sizes.h"
#include "kernels.cuh"
#include "timer.h"

namespace cuda
{
    void matmul(float* A, float* B, float* C, int rows_A, int cols_A, int cols_B)
    {
        Info info = getInfo();

        float *dA, *dB, *dC;

        hipMalloc(&dA, SIZE_A_BYTES);
        hipMalloc(&dB, SIZE_B_BYTES);
        hipMalloc(&dC, SIZE_C_BYTES);

        {
            ScopedTimer timer("A,B to device", POST);
            hipMemcpy(dA, A, SIZE_A_BYTES, hipMemcpyHostToDevice);
            hipMemcpy(dB, B, SIZE_B_BYTES, hipMemcpyHostToDevice);
        }

        {
            ScopedTimer timer("matmul kernel", POST);

            dim3 tiles(CEIL_DIV(COLS_C, tileDim.x), CEIL_DIV(ROWS_C, tileDim.y));

            dim3 gridDim = tiles;
            dim3 blockDim = tileDim;
            int sharedMemSize = 2 * dim2ToBytes(tileDim);

            // print info

            printf("gridDim: %d %d %d\n", gridDim.x, gridDim.y, gridDim.z);
            printf("blockDim: %d %d %d\n", blockDim.x, blockDim.y, blockDim.z);
            std::cout << "sharedMemSize per block: " << humanReadableMemSize(sharedMemSize) << " / " << humanReadableMemSize(info.sharedMemPerBlock) << std::endl;
            std::cout << "sharedMemSize per grid: " << humanReadableMemSize(sharedMemSize * gridDim.x * gridDim.y * gridDim.z) << " / " << humanReadableMemSize(info.sharedMemPerMultiprocessor * info.multiProcessorCount) << std::endl;

            kernels::tiled_matmul<<<gridDim, blockDim, sharedMemSize>>>(dA, dB, dC, rows_A, cols_A, cols_B);

            hipDeviceSynchronize();
            CUDA_CHECK
        }

        {
            ScopedTimer timer("C to host", POST);
            hipMemcpy(C, dC, SIZE_C_BYTES, hipMemcpyDeviceToHost);
        }

        hipFree(dA);
        hipFree(dB);
        hipFree(dC);
    }
}
