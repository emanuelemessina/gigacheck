#include "inferred_matrix_sizes.h"
#include "kernels.cuh"
#include "timer.h"

namespace cuda
{
    void matmul(float* A, float* B, float* C, int rows_A, int cols_A, int cols_B)
    {
        float *dA, *dB, *dC;

        hipMalloc(&dA, SIZE_A_BYTES);
        hipMalloc(&dB, SIZE_B_BYTES);
        hipMalloc(&dC, SIZE_C_BYTES);

        {
            ScopedTimer timer("A,B to device", POST);
            hipMemcpy(dA, A, SIZE_A_BYTES, hipMemcpyHostToDevice);
            hipMemcpy(dB, B, SIZE_B_BYTES, hipMemcpyHostToDevice);
        }

        {
            ScopedTimer timer("matmul kernel", POST);

            dim3 tiles(CEIL_DIV(COLS_C, tileDim.x), CEIL_DIV(ROWS_C, tileDim.y));

            dim3 gridSize = tiles;
            dim3 blockSize = tileDim;
            int sharedMemSize = 2 * dim2ToBytes(blockSize);

            kernels::tiled_matmul<<<gridSize, blockSize, sharedMemSize>>>(dA, dB, dC, rows_A, cols_A, cols_B);

            hipDeviceSynchronize();
            CUDA_CHECK
        }

        {
            ScopedTimer timer("dC to host", POST);
            hipMemcpy(C, dC, SIZE_C_BYTES, hipMemcpyDeviceToHost);
        }

        hipFree(dA);
        hipFree(dB);
        hipFree(dC);
    }
}