#include "hip/hip_runtime.h"
#include "cuda.cuh"
#include "edc.cuh"
#include "globals.h"
#include "inferred_matrix_sizes.h"
#include "kernels.cuh"
#include "matrix.h"
#include "timer.h"
#include <vector>

#define CUDA_DEBUG_PRINT 0

#define SWAP(a, b)    \
    {                 \
        auto tmp = a; \
        a = b;        \
        b = tmp;      \
    }

enum class OperandMatrix
{
    A = 'A',
    B = 'B'
};

/**
 * @brief Copies a matrix (or a portion of it) to CUDA memory
 *
 * It is able to handle also the copy of a block of a matrix.
 *
 * Assume we have to copy block (i, j) of a matrix HxW, divided into NxM blocks.
 *
 * The block (therefore also the GPU matrix) will have H/N rows and W/M cols.
 *
 * To copy the correct block, we have to start at an initial offset that includes:
 * - a delta of i * W * H/N, to select the correct row of blocks
 * - an extra delta of j * W/M to select the correct block within the row
 *
 * Moreover, if  M != 1 the next row of the matrix is not immediately after the previous one,
 * but it starts W cells after the previous one starts (next_row_offset)
 *
 * @param[in]   matrix                  The original, host matrix
 * @param[out]  dst                     The GPU allocated memory where to copy
 * @param[in]   rows                    The number of rows that should be copied
 * @param[in]   cols                    The number of columns that should be copied
 * @param[in]   allocated_cols          The number of columns that are allocated for each matrix row (may be an overallocation of cols)
 * @param[in]   initial_offset          How much of the original matrix must be skipped at the beginning
 * @param[in]   next_row_offset         How much of the original matrix must be skipped to transition to the next row
 * @param[in]   will_need_row_checksum  Whether to copy this block leaving a free column in device memory to store the row checksum vector
 * @param[in]   stream                  on which to run the async copy
 */
void host_block_to_device(float* matrix, float* dst, int rows, int cols, int allocated_cols, int initial_offset, int next_row_offset, bool will_need_row_checksum, hipStream_t stream)
{
    matrix += initial_offset;
    for (int i = 0; i < rows; i++)
    {
        hipMemcpyAsync(dst, matrix, cols * sizeof(float), hipMemcpyHostToDevice, stream);

        matrix += next_row_offset;
        dst += allocated_cols + (will_need_row_checksum ? 1 : 0);
    }
}

/**
 * @brief The reverse of host_block_to_device
 *
 * @param[in]   matrix                The original, GPU matrix
 * @param[out]  dst                   The host memory allocated memory where to copy
 * @param[in]   rows                  The number of rows that should be copied
 * @param[in]   cols                  The number of columns that should be copied
 * @param[in]   allocated_cols        The number of columns that are allocated for each matrix row (may be an overallocation of cols)
 * @param[in]   initial_offset        How much of the host matrix must be skipped at the beginning
 * @param[in]   next_row_offset       How much of the host matrix must be skipped to transition to the next row
 * @param[in]   stream                which to run the async copy
 */
void device_block_to_host(float* matrix, float* dst, int rows, int cols, int allocated_cols, int initial_offset, int next_row_offset, hipStream_t stream)
{
    dst += initial_offset;
    for (int i = 0; i < rows; i++)
    {
        hipMemcpyAsync(dst, matrix, cols * sizeof(float), hipMemcpyDeviceToHost, stream);

        dst += next_row_offset;
        matrix += allocated_cols;
    }
}

/**
 * @brief Loads a host matrix block to GPU memory, optionally computing a row/col checksum
 *
 * @param[in]   h_mat             src host matrix
 * @param[out]  d_mat             dst device matrix
 * @param[in]   rows           Total number of rows in the matrix
 * @param[in]   cols           Total number of cols in the matrix
 * @param[in]   num_split_row     The number of blocks a row was split into
 * @param[in]   num_split_col     The number of blocks a col was split into
 * @param[in]   max_block_rows    The max amount of rows of a block
 * @param[in]   max_block_cols    The max amount of cols of a block
 * @param[in]   block_idy         Vertical index of the block to be copied
 * @param[in]   block_idx         Horizontal index of the block to be copied
 * @param[in]   stream            Which stream to use for the async operations
 *
 */
void loadcheck_input_block(OperandMatrix operand, float* h_mat, float* d_mat, int rows, int cols, int num_split_row, int num_split_col, int max_block_rows, int max_block_cols, int block_idy, int block_idx, hipStream_t stream)
{
    // copy to device
    int extra = globals::noEDC ? 0 : 1;
    int size = operand == OperandMatrix::A ? ((max_block_rows + extra) * max_block_cols * sizeof(float)) : (max_block_rows * (max_block_cols + extra) * sizeof(float));
    if (block_idx == num_split_col - 1 || block_idy == num_split_row - 1)
        hipMemsetAsync(d_mat, 0, size, stream);

    int block_rows = CEIL_DIV(rows, num_split_row);
    int block_cols = CEIL_DIV(cols, num_split_col);
    int offset = block_idy * max_block_rows * cols + block_idx * max_block_cols;

    if (block_idx == num_split_col - 1)
        block_cols = cols - block_cols * block_idx;
    if (block_idy == num_split_row - 1)
        block_rows = rows - block_rows * block_idy;

    host_block_to_device(h_mat, d_mat, block_rows, block_cols, max_block_cols, offset, cols, operand == OperandMatrix::B && !globals::noEDC, stream);

    if (!globals::noEDC)
    {
        // calculate col checksum for A / row checksum for B
        dim3 gridDim = operand == OperandMatrix::A ? dim3(max_block_cols) : dim3(1, max_block_rows);
        dim3 blockDim = operand == OperandMatrix::A ? dim3(1, tileDim.y) : dim3(tileDim.x, 1);
        int sharedMemSize = linearDimToBytes(operand == OperandMatrix::A ? tileDim.y : tileDim.x);
        ReductionDirection direction = operand == OperandMatrix::A ? ReductionDirection::ALONG_COL : ReductionDirection::ALONG_ROW;
        kernels::compute_checksums<<<gridDim, blockDim, sharedMemSize, stream>>>(d_mat, max_block_rows, max_block_cols, direction);

        std::pair<uint64_t, uint64_t> m = kernels::metrics::compute_checksums(dimsToN(gridDim, blockDim), max(blockDim.x, blockDim.y), direction, false);
        globals::profiling::flop_counter += m.first;
        globals::profiling::transfer_counter += m.second;
    }

    // Print mat (with checksums)
    if (globals::debugPrint)
    {
        std::string name(1, (char)operand);
        int flags = IS_DEVICE_MAT;

        if (!globals::noEDC)
        {
            name += " (w/ ";
            name += operand == OperandMatrix::A ? "column" : "row";
            name += " checksum)";

            flags |= operand == OperandMatrix::A ? HIGHLIGHT_LAST_ROW : HIGHLIGHT_LAST_COL;
        }

        matrix::print(
            d_mat,
            max_block_rows + (operand == OperandMatrix::A ? extra : 0),
            max_block_cols + (operand == OperandMatrix::A ? 0 : extra),
            name.c_str(),
            flags);
    }
}

/**
 * @brief Compute the checksum for matrix C
 *
 * @param[inout]  C                 The matrix on which to compute the checksums
 * @param[in]     direction         Whether to compute row or col checksums
 * @param[in]     max_block_cols_B  The max amount of cols in any block of B
 * @param[in]     max_block_rows_A  The max amount of rows in any block of A
 * @param[in]     stream            Which stream to use for the async operations
 * @param[out]    result_array      The array where to put the checksums. If NULL, they are inserted in the last row/col of C
 *
 */
void compute_control_checksums(float* C, ReductionDirection direction, int max_block_cols_B, int max_block_rows_A, hipStream_t stream, float* result_array)
{
    // compute col control checksum

    if (direction == ReductionDirection::ALONG_COL)
    {
        dim3 gridDim = dim3(MAX_BLOCK_COLS_C + 1);
        dim3 blockDim = dim3(1, tileDim.y);
        int sharedMemSize = linearDimToBytes(tileDim.y);
        kernels::compute_checksums<<<gridDim, blockDim, sharedMemSize, stream>>>(C, MAX_BLOCK_ROWS_C, (MAX_BLOCK_COLS_C + 1), ReductionDirection::ALONG_COL, result_array);

        std::pair<uint64_t, uint64_t> m = kernels::metrics::compute_checksums(dimsToN(gridDim, blockDim), max(blockDim.x, blockDim.y), ReductionDirection::ALONG_COL, true);
        globals::profiling::flop_counter += m.first;
        globals::profiling::transfer_counter += m.second;
    }

    // compute row control checksum
    else
    {
        dim3 gridDim = dim3(1, MAX_BLOCK_ROWS_C + 1);
        dim3 blockDim = dim3(tileDim.x, 1);
        int sharedMemSize = linearDimToBytes(tileDim.x);
        kernels::compute_checksums<<<gridDim, blockDim, sharedMemSize, stream>>>(C, (MAX_BLOCK_ROWS_C + 1), MAX_BLOCK_COLS_C, ReductionDirection::ALONG_ROW, result_array);

        std::pair<uint64_t, uint64_t> m = kernels::metrics::compute_checksums(dimsToN(gridDim, blockDim), max(blockDim.x, blockDim.y), ReductionDirection::ALONG_ROW, true);
        globals::profiling::flop_counter += m.first;
        globals::profiling::transfer_counter += m.second;
    }
}

namespace cuda
{
    /**
     * @brief Executes multiplication, error injection, error detection and error correction
     *
     * @param[in]   A                     The first matrix to multiply
     * @param[in]   B                     The second matrix to multiply
     * @param[out]  C                     The result matrix
     * @param[in]   rows_A                Total number of rows in A
     * @param[in]   cols_B                Total number of cols in B
     * @param[in]   num_split_common_dim  The number of blocks the matrix was split into (in the direction where blocks of A and B must have the same size)
     * @param[in]   num_split_other_dim   The number of blocks the matrix was split into (in the other direction)
     * @param[in]   max_block_rows_A      The max amount of rows in any block of A
     * @param[in]   max_block_cols_A      The max amount of cols in any block of A
     * @param[in]   max_block_cols_B      The max amount of cols in any block of B
     * @param[in]   C_block_idy                 The coordinates (row) of the block of C that is being calculated
     * @param[in]   C_block_idx                 The coordinates (col) of the block of C that is being calculated
     * @param[out]  block_rows_C_cur      The number of meaningful rows in the current block of C
     * @param[out]  block_cols_C_cur      The number of meaningful cols in the current block of C
     * @param[in]   stream                Which stream to use as main stream
     * @param[in]   streamBis             Which stream to use for operations concurrent to the main stream
     * @param[in]   errors_count          The number of errors to be introduced
     * @param[in]   error_xs              The coordinates (x) of the errors to be introduced
     * @param[in]   error_ys              The coordinates (y) of the errors to be introduced
     * @param[in]   error_values          The values of the errors
     * @param[out]  result_correct        Whether the final value of C left in output is correct
     * @param[out]  result_corrected      Whether the matrix had errors, but all errors were corrected
     *
     */
    void mul_inject_edc(float* A,
                        float* B,
                        float* C,
                        int rows_A,
                        int cols_B,
                        int num_split_common_dim,
                        int num_split_other_dim,
                        int max_block_rows_A,
                        int max_block_cols_A,
                        int max_block_cols_B,
                        int C_block_idy,
                        int C_block_idx,
                        int* block_rows_C_cur,
                        int* block_cols_C_cur,
                        hipStream_t stream,
                        hipStream_t streamBis,
                        int errors_count,
                        int* error_xs,
                        int* error_ys,
                        float* error_values,
                        bool* result_correct,
                        bool* result_corrected)
    {
        if (globals::debugPrint)
        {
            matrix::print(C, MAX_BLOCK_ROWS_C + 1, MAX_BLOCK_COLS_C + 1, "C (pre mul)", HIGHLIGHT_LAST_ROW_AND_COL | IS_DEVICE_MAT);
        }

        int extra = globals::noEDC ? 0 : 1;

        // rows, cols for dC
        (*block_rows_C_cur) = CEIL_DIV(ROWS_C, num_split_other_dim);
        (*block_cols_C_cur) = CEIL_DIV(COLS_C, num_split_other_dim);

        if (C_block_idy == num_split_other_dim - 1)
            (*block_rows_C_cur) = ROWS_C - (*block_rows_C_cur) * C_block_idy;
        if (C_block_idx == num_split_other_dim - 1)
            (*block_cols_C_cur) = COLS_C - (*block_cols_C_cur) * C_block_idx;

        // compute the actual matrix multiplication as usual

        {
            dim3 gridDim = dim3(CEIL_DIV(MAX_BLOCK_COLS_C + extra, tileDim.x), CEIL_DIV(MAX_BLOCK_ROWS_C + extra, tileDim.y));
            int sharedMemSize = 2 * dim2ToBytes(tileDim);
            kernels::tiled_matmul<<<gridDim, tileDim, sharedMemSize, stream>>>(A, B, C, max_block_rows_A + extra, max_block_cols_A, max_block_cols_B + extra);

            CUDA_CHECK

            std::pair<uint64_t, uint64_t> m = kernels::metrics::tiled_matmul(dimsToN(gridDim, tileDim), tileDim.x);
            globals::profiling::flop_counter += m.first;
            globals::profiling::transfer_counter += m.second;
        }

        if (globals::debugPrint)
        {
            hipStreamSynchronize(stream);
            matrix::print(C, MAX_BLOCK_ROWS_C + 1, MAX_BLOCK_COLS_C + 1, "C", HIGHLIGHT_LAST_ROW_AND_COL | IS_DEVICE_MAT);
        }

        if (globals::noEDC)
            return;

        float *d_cc_control, *d_rc_control;
        hipMalloc(&d_cc_control, (MAX_BLOCK_COLS_C + 1) * sizeof(float));
        hipMalloc(&d_rc_control, (MAX_BLOCK_ROWS_C + 1) * sizeof(float));

        hipEvent_t C_err_added;
        hipEventCreate(&C_err_added);

        // introduce errors in dC
        {
            for (int i = 0; i < errors_count; i++)
            {
                float tmp;
                hipMemcpyAsync(&tmp, C + error_ys[i] * (MAX_BLOCK_COLS_C + 1) + error_xs[i], sizeof(float), hipMemcpyDeviceToHost, stream);
                hipStreamSynchronize(stream);
                tmp += error_values[i];
                hipMemcpyAsync(C + error_ys[i] * (MAX_BLOCK_COLS_C + 1) + error_xs[i], &tmp, sizeof(float), hipMemcpyHostToDevice, stream);
                hipStreamSynchronize(stream);
            }
            hipEventRecord(C_err_added, stream);

            CUDA_CHECK
        }

#if CUDA_DEBUG_PRINT
        printf("error xs: ");
        for (int i = 0; i < errors_count; i++)
            printf("%d ", error_xs[i]);
        printf("\n");
        printf("error ys: ");
        for (int i = 0; i < errors_count; i++)
            printf("%d ", error_ys[i]);
        printf("\n");
        printf("error vs: ");
        for (int i = 0; i < errors_count; i++)
            printf("%f ", error_values[i]);
        printf("\n");
#endif

        // print dC (with mul checksums)
        if (globals::debugPrint)
        {
            hipStreamSynchronize(stream);
            matrix::print(C, MAX_BLOCK_ROWS_C + 1, MAX_BLOCK_COLS_C + 1, "C (w/ errors)", HIGHLIGHT_LAST_ROW_AND_COL | IS_DEVICE_MAT, error_xs, error_ys, errors_count);
        }

        // compute control checksums after mul
        {
            // compute col control checksum
            compute_control_checksums(C, ReductionDirection::ALONG_COL, max_block_cols_B, max_block_rows_A, stream, d_cc_control);

            // compute row control checksum
            hipStreamWaitEvent(streamBis, C_err_added);
            compute_control_checksums(C, ReductionDirection::ALONG_ROW, max_block_cols_B, max_block_rows_A, streamBis, d_rc_control);

            CUDA_CHECK
        }

        // print control checksums
        if (globals::debugPrint)
        {
            hipStreamSynchronize(stream);
            hipStreamSynchronize(streamBis);
            std::vector<int> zeros(errors_count, 0);
            matrix::print(d_rc_control, MAX_BLOCK_ROWS_C + 1, 1, "C control row checksum", HIGHLIGHT_LAST_COL | IS_DEVICE_MAT, zeros.data(), error_ys, errors_count);
            matrix::print(d_cc_control, 1, MAX_BLOCK_COLS_C + 1, "C control column checksum", HIGHLIGHT_LAST_ROW | IS_DEVICE_MAT, error_xs, zeros.data(), errors_count);
        }

        // edc

        {
            bool recompute_vertical_checksums = false;
            bool recompute_horizontal_checksums = false;

            EDCResult edc_res = errors_detect_correct(C, MAX_BLOCK_ROWS_C, MAX_BLOCK_COLS_C, d_cc_control, d_rc_control, stream, streamBis, &recompute_vertical_checksums, &recompute_horizontal_checksums);

            // choice: don't send back the result if it's wrong
            // NOTE: now the result may be partial, since an error will stop the rest
            switch (edc_res)
            {
                case UNCORRECTABLE_ERROR:
                    *result_correct = false;
                    break;

                case CORRECTED_ERROR:
                    *result_corrected = true;
                    break;

                case NO_ERROR:
                    break;
            }

            if (recompute_horizontal_checksums)
                compute_control_checksums(C, ReductionDirection::ALONG_ROW, max_block_cols_B, max_block_rows_A, stream, NULL);

            if (recompute_vertical_checksums)
                compute_control_checksums(C, ReductionDirection::ALONG_COL, max_block_cols_B, max_block_rows_A, stream, NULL);
        }

        hipFree(d_cc_control);
        hipFree(d_rc_control);
        hipEventDestroy(C_err_added);
    }

    EDCResult matmul_ec(float* A, float* B, float* C, int rows_A, int cols_A, int cols_B, int errors_count, int** per_block_error_xs, int** per_block_error_ys, float** error_values, MulStrategy strategy)
    {
        // calculate the number of blocks to split A and B into, based on the chosen strategy and the available memory

        int num_split_common_dim;
        int num_split_other_dim;

        if (!matrix::calc_splits(strategy, rows_A, cols_A, cols_B, &num_split_common_dim, &num_split_other_dim))
        {
            CERR << "Not enough device memory to store the checksums, aborting." << ENDL;
            return NO_ERROR;
        }

        // calc block dimensions based on the number of splits

        int max_block_rows_A = CEIL_DIV(rows_A, num_split_other_dim);
        int max_block_cols_A = CEIL_DIV(cols_A, num_split_common_dim);
        int max_block_cols_B = CEIL_DIV(cols_B, num_split_other_dim);

        // register host pointers as pinned

        hipHostRegister(A, SIZE_A_BYTES, hipHostRegisterDefault);
        hipHostRegister(B, SIZE_B_BYTES, hipHostRegisterDefault);
        hipHostRegister(C, SIZE_C_BYTES, hipHostRegisterDefault);

        CUDA_CHECK

        // allocate device matrices with extra space for checksums A(m+1xn)B(nxp+1) = C(m+1xp+1)

        int extra = globals::noEDC ? 0 : 1;
        int size_A_ec = (max_block_rows_A + extra) * max_block_cols_A * sizeof(float);
        int size_B_ec = MAX_BLOCK_ROWS_B * (max_block_cols_B + extra) * sizeof(float);
        int size_C_ec = (MAX_BLOCK_ROWS_C + extra) * (MAX_BLOCK_COLS_C + extra) * sizeof(float);

        float *dA, *dB, *dC, *dA_alt, *dB_alt, *dC_alt; // declare device pointers for every strategy

        switch (strategy)
        {
            case preloadAB_deferUnloadC:        // AB->C, A'B', C' (while mul on C/C', store C'/C, alternating result buffer strategy)
            case parallelMul:                   // AB->C, A'B'->C' (two muls in parallel, two result buffers)
                hipMalloc(&dC_alt, size_C_ec); // in both cases we need a buffer for C'

            case preloadAB:                         // AB->C, A'B' (while mul on C, load next blocks A' and B')
                if (strategy != parallelMul)        // exploit common A in parallel mul, don't allocate A'
                    hipMalloc(&dA_alt, size_A_ec); // buffer for A'
                hipMalloc(&dB_alt, size_B_ec);     // buffer for B'

            case simple: // base buffers AB->C
                hipMalloc(&dA, size_A_ec);
                hipMalloc(&dB, size_B_ec);
                hipMalloc(&dC, size_C_ec);
        }

        CUDA_CHECK

        // declare streams for parallel executions

        hipStream_t stream_A;
        hipStream_t stream_B;
        hipStream_t stream_C;
        hipStream_t stream_Cbis;

        hipStream_t stream_A_alt;
        hipStream_t stream_B_alt;
        hipStream_t stream_C_alt;
        hipStream_t stream_Cbis_alt;

        switch (strategy) // create just the streams we need
        {
            case preloadAB_deferUnloadC:
            case parallelMul:
                hipStreamCreate(&stream_C_alt);
                hipStreamCreate(&stream_Cbis_alt);

            case preloadAB:
                hipStreamCreate(&stream_A_alt);
                hipStreamCreate(&stream_B_alt);

            case simple:
                hipStreamCreate(&stream_A);
                hipStreamCreate(&stream_B);
                hipStreamCreate(&stream_C);
                hipStreamCreate(&stream_Cbis);
        }

        // declare events

        hipEvent_t A_copied;
        hipEvent_t B_copied;
        hipEvent_t A_alt_copied;
        hipEvent_t B_alt_copied;

        hipEvent_t A_can_be_overwritten;
        hipEvent_t B_can_be_overwritten;
        hipEvent_t B_alt_can_be_overwritten;

        // block result flags

        bool result_correct = true;
        bool result_corrected = false;
        bool result_correct_alt = true;
        bool result_corrected_alt = false;

        // actual block dimensions placeholders

        int block_rows_C_cur;
        int block_cols_C_cur;
        int block_rows_C_alt;
        int block_cols_C_alt;

        // init events
        CUDA_CREATE_RECORD_EVENT(A_can_be_overwritten, stream_A);
        CUDA_CREATE_RECORD_EVENT(B_can_be_overwritten, stream_B);
        if (strategy == parallelMul)
            CUDA_CREATE_RECORD_EVENT(B_alt_can_be_overwritten, stream_B_alt);

        if (strategy == preloadAB || strategy == preloadAB_deferUnloadC) // when using preloading, we need to load the first two operand blocks
        {
            loadcheck_input_block(OperandMatrix::A, A, dA, rows_A, cols_A, num_split_other_dim, num_split_common_dim, max_block_rows_A, max_block_cols_A, 0, 0, stream_A);
            CUDA_CREATE_RECORD_EVENT(A_copied, stream_A);

            loadcheck_input_block(OperandMatrix::B, B, dB, ROWS_B, cols_B, num_split_common_dim, num_split_other_dim, MAX_BLOCK_ROWS_B, max_block_cols_B, 0, 0, stream_B);
            CUDA_CREATE_RECORD_EVENT(B_copied, stream_B);
        }

        // multiply in blocks

        for (int C_block_idy = 0; C_block_idy < num_split_other_dim && result_correct; C_block_idy++) // iterate over C blocks vertically
        {
            for (int C_block_idx = 0; C_block_idx < num_split_other_dim && result_correct; C_block_idx += (strategy == parallelMul ? 2 : 1)) // iterate over C blocks horizontally (if 2 muls we process two cols at a time)
            {
                // clear the result buffer(s) so we can perform additive mul with one kernel
                hipMemsetAsync(dC, 0, size_C_ec, stream_C);
                if (strategy == parallelMul && C_block_idx + 1 < num_split_other_dim)
                    hipMemsetAsync(dC_alt, 0, size_C_ec, stream_C_alt);

                for (int block_common_id = 0; block_common_id < num_split_common_dim && result_correct; block_common_id++) // iterate over blocks along the common dimension
                {

                    /*
                    Note on sending matrices to device and calculate checksums in parallel on different streams
                        stream1: copyA, checkA
                        stream2: copyB, checkB
                        copies are sent to possibly the same copy queue, kernels to the same kernel queue (especially if only one queue per category exists)
                        we use depth-first issue order: copyA - checkA - copyB - checkB
                        breadth-first issue order would be: copyA - copyB - checkA - checkB
                        we found that depth-first gives better performance
                        loadcheck order is: copy, check
                    */

                    {
                        if (strategy == simple || strategy == parallelMul) // load the base blocks A,B
                        {
                            CUDA_WAIT_EVENT_DESTROY(A_can_be_overwritten, stream_A)
                            loadcheck_input_block(OperandMatrix::A, A, dA, rows_A, cols_A, num_split_other_dim, num_split_common_dim, max_block_rows_A, max_block_cols_A, C_block_idy, block_common_id, stream_A);
                            CUDA_CREATE_RECORD_EVENT(A_copied, stream_A);

                            CUDA_WAIT_EVENT_DESTROY(B_can_be_overwritten, stream_B)
                            loadcheck_input_block(OperandMatrix::B, B, dB, ROWS_B, cols_B, num_split_common_dim, num_split_other_dim, MAX_BLOCK_ROWS_B, max_block_cols_B, block_common_id, C_block_idx, stream_B);
                            CUDA_CREATE_RECORD_EVENT(B_copied, stream_B);
                        }
                        if (strategy == parallelMul) // load B' for parallel mul
                        {
                            if (C_block_idx + 1 < num_split_other_dim)
                            {
                                CUDA_WAIT_EVENT_DESTROY(B_alt_can_be_overwritten, stream_B_alt)
                                loadcheck_input_block(OperandMatrix::B, B, dB_alt, ROWS_B, cols_B, num_split_common_dim, num_split_other_dim, MAX_BLOCK_ROWS_B, max_block_cols_B, block_common_id, C_block_idx + 1, stream_B_alt);
                                CUDA_CREATE_RECORD_EVENT(B_alt_copied, stream_B_alt);
                            }
                        }
                        if (strategy == preloadAB || strategy == preloadAB_deferUnloadC) // preload A',B'
                        {
                            // if strategy pre-loads A and B, and this is not the last iteration, pre-load the next A, B
                            if (block_common_id != (num_split_common_dim - 1) || C_block_idy != (num_split_other_dim - 1) || C_block_idx != (num_split_other_dim - 1))
                            {
                                int next_block = block_common_id + 1;
                                int next_C_block_idx = C_block_idx;
                                int next_C_block_idy = C_block_idy;
                                if (next_block == num_split_common_dim)
                                {
                                    next_block = 0;
                                    next_C_block_idx = C_block_idx + 1;
                                    if (next_C_block_idx == num_split_other_dim)
                                    {
                                        next_C_block_idx = 0;
                                        next_C_block_idy = C_block_idy + 1;
                                    }
                                }

                                CUDA_WAIT_EVENT_DESTROY(A_can_be_overwritten, stream_A_alt)
                                loadcheck_input_block(OperandMatrix::A, A, dA_alt, rows_A, cols_A, num_split_other_dim, num_split_common_dim, max_block_rows_A, max_block_cols_A, next_C_block_idy, next_block, stream_A_alt);
                                CUDA_CREATE_RECORD_EVENT(A_alt_copied, stream_A_alt);

                                CUDA_WAIT_EVENT_DESTROY(B_can_be_overwritten, stream_B_alt)
                                loadcheck_input_block(OperandMatrix::B, B, dB_alt, ROWS_B, cols_B, num_split_common_dim, num_split_other_dim, MAX_BLOCK_ROWS_B, max_block_cols_B, next_block, next_C_block_idx, stream_B_alt);
                                CUDA_CREATE_RECORD_EVENT(B_alt_copied, stream_B_alt);
                            }
                        }

                        CUDA_CHECK
                    }

                    int block_id = block_common_id + C_block_idx * num_split_common_dim + C_block_idy * num_split_common_dim * num_split_other_dim;

                    // Wait for operands to be loaded
                    CUDA_WAIT_EVENT_DESTROY_IF(A_copied, stream_C, strategy != parallelMul || C_block_idx + 1 >= num_split_other_dim)
                    CUDA_WAIT_EVENT_DESTROY(B_copied, stream_C)

                    // Compute product
                    mul_inject_edc(dA, dB, dC, rows_A, cols_B, num_split_common_dim, num_split_other_dim, max_block_rows_A, max_block_cols_A, max_block_cols_B, C_block_idy, C_block_idx, &block_rows_C_cur, &block_cols_C_cur, stream_C, stream_Cbis, errors_count, per_block_error_xs[block_id], per_block_error_ys[block_id], error_values[block_id], &result_correct, &result_corrected);

                    // Notify that A and B are no longer needed, and as such they can be overwritten
                    if (strategy != parallelMul || C_block_idx + 1 >= num_split_other_dim)
                        CUDA_CREATE_RECORD_EVENT(A_can_be_overwritten, stream_C);
                    CUDA_CREATE_RECORD_EVENT(B_can_be_overwritten, stream_C);

                    // If parallel multiplication, and C' is meaningful, compute the other product
                    if (strategy == parallelMul && C_block_idx + 1 < num_split_other_dim)
                    {
                        CUDA_WAIT_EVENT_DESTROY(A_copied, stream_C_alt)
                        CUDA_WAIT_EVENT_DESTROY(B_alt_copied, stream_C_alt)

                        block_id = block_common_id + (C_block_idx + 1) * num_split_common_dim + C_block_idy * num_split_common_dim * num_split_other_dim;
                        mul_inject_edc(dA, dB_alt, dC_alt, rows_A, cols_B, num_split_common_dim, num_split_other_dim, max_block_rows_A, max_block_cols_A, max_block_cols_B, C_block_idy, C_block_idx + 1, &block_rows_C_alt, &block_cols_C_alt, stream_C_alt, stream_Cbis_alt, errors_count, per_block_error_xs[block_id], per_block_error_ys[block_id], error_values[block_id], &result_correct_alt, &result_corrected_alt);

                        CUDA_CREATE_RECORD_EVENT(A_can_be_overwritten, stream_C);
                        CUDA_CREATE_RECORD_EVENT(B_alt_can_be_overwritten, stream_C_alt);
                    }

                    // Switch A and B buffers if required
                    switch (strategy)
                    {
                        case preloadAB_deferUnloadC:
                        case preloadAB:
                            SWAP(dA, dA_alt)
                            SWAP(dB, dB_alt)
                            SWAP(stream_A, stream_A_alt)
                            SWAP(stream_B, stream_B_alt)
                            SWAP(A_copied, A_alt_copied)
                            SWAP(B_copied, B_alt_copied)
                    }
                }

                // result block has been accumulated
                // send it to host mem (without checksums)
                // (send two blocks in case of parallel mul)

                {
                    int offset = C_block_idy * MAX_BLOCK_ROWS_C * COLS_C + C_block_idx * MAX_BLOCK_COLS_C;
                    int offset2 = C_block_idy * MAX_BLOCK_ROWS_C * COLS_C + (C_block_idx + 1) * MAX_BLOCK_COLS_C; // for parallel mul

                    switch (strategy)
                    {
                        case preloadAB_deferUnloadC:
                            SWAP(dC, dC_alt)
                            SWAP(stream_C, stream_C_alt)
                            SWAP(stream_Cbis, stream_Cbis_alt)
                            SWAP(block_rows_C_cur, block_rows_C_alt)
                            SWAP(block_cols_C_cur, block_cols_C_alt)

                            device_block_to_host(dC_alt, C, block_rows_C_alt, block_cols_C_alt, MAX_BLOCK_COLS_C + extra, offset, COLS_C, stream_C_alt);
                            break;

                        case parallelMul:
                            if (C_block_idx + 1 < num_split_other_dim)
                                device_block_to_host(dC_alt, C, block_rows_C_alt, block_cols_C_alt, MAX_BLOCK_COLS_C + extra, offset2, COLS_C, stream_C_alt);

                        case preloadAB:
                        case simple:
                            device_block_to_host(dC, C, block_rows_C_cur, block_cols_C_cur, MAX_BLOCK_COLS_C + extra, offset, COLS_C, stream_C);
                    }

                    CUDA_CHECK
                }
            }
        }

        // cleanup:

        {
            switch (strategy)
            {
                case preloadAB_deferUnloadC:
                case parallelMul:
                    hipStreamDestroy(stream_C_alt);
                    hipStreamDestroy(stream_Cbis_alt);

                case preloadAB:
                    hipStreamDestroy(stream_A_alt);
                    hipStreamDestroy(stream_B_alt);

                case simple:
                    hipStreamDestroy(stream_A);
                    hipStreamDestroy(stream_B);
                    hipStreamDestroy(stream_C);
                    hipStreamDestroy(stream_Cbis);
            }

            switch (strategy)
            {
                case preloadAB_deferUnloadC:
                case parallelMul:
                    hipFree(dC_alt);

                case preloadAB:
                    if (strategy != parallelMul)
                        hipFree(dA_alt);
                    hipFree(dB_alt);

                case simple:
                    hipFree(dA);
                    hipFree(dB);
                    hipFree(dC);
            }

            hipHostUnregister(A);
            hipHostUnregister(B);
            hipHostUnregister(C);

            CUDA_CHECK
        }

        if (!result_correct)
            return UNCORRECTABLE_ERROR;
        return result_corrected ? CORRECTED_ERROR : NO_ERROR;
    }
}
