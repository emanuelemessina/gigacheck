#include "cuda.cuh"
#include "edc.cuh"
#include "globals.h"
#include "inferred_matrix_sizes.h"
#include "kernels.cuh"
#include "matrix.h"
#include "timer.h"

#define SWAP(a, b)    \
    {                 \
        auto tmp = a; \
        a = b;        \
        b = tmp;      \
    }

#define CUDA_WAIT_DESTROY_EVENT(event, stream, destroy) \
    {                                                   \
        hipStreamWaitEvent(stream, event);             \
        if (destroy)                                    \
            hipEventDestroy(event);                    \
    }

#define CUDA_CREATE_RECORD_EVENT(event, stream) \
    {                                           \
        hipEventCreate(&event);                \
        hipEventRecord(event, stream);         \
    }

/**
 * @brief Copies a matrix (or a portion of it) to CUDA memory
 *
 * It is able to handle also the copy of a block of a matrix.
 *
 * Assume we have to copy block (i, j) of a matrix HxW, divided into NxM blocks.
 *
 * The block (therefore also the GPU matrix) will have H/N rows and W/M cols.
 *
 * To copy the correct block, we have to start at an initial offset that includes:
 * - a delta of i * W * H/N, to select the correct row of blocks
 * - an extra delta of j * W/M to select the correct block within the row
 *
 * Moreover, if  M != 1 the next row of the matrix is not immediately after the previous one,
 * but it starts W cells after the previous one starts (next_row_offset)
 *
 * @param[in]   matrix                The original, host matrix
 * @param[out]  dst                   The GPU allocated memory where to copy
 * @param[in]   rows                  The number of rows that should be copied
 * @param[in]   cols                  The number of columns that should be copied
 * @param[in]   allocated_cols        The number of columns that are allocated for each matrix row (may be an overallocation of cols)
 * @param[in]   initial_offset        How much of the original matrix must be skipped at the beginning
 * @param[in]   next_row_offset       How much of the original matrix must be skipped to transition to the next row
 * @param[in]   leave_cell_after_row  If a cell should be left empty after each row (== copying B or a block of B)
 * @param[in]   stream                The stream on which to work
 */
void cp_matrix_to_CUDA(float* matrix, float* dst, int rows, int cols, int allocated_cols, int initial_offset, int next_row_offset, bool leave_cell_after_row, hipStream_t stream)
{
    matrix += initial_offset;
    for (int i = 0; i < rows; i++)
    {
        hipMemcpyAsync(dst, matrix, cols * sizeof(float), hipMemcpyHostToDevice, stream);

        matrix += next_row_offset;
        dst += allocated_cols + (leave_cell_after_row ? 1 : 0);
    }
}

/**
 * @brief The reverse of cp_matrix_to_CUDA
 *
 * @param[in]   matrix                The original, GPU matrix
 * @param[out]  dst                   The host memory allocated memory where to copy
 * @param[in]   rows                  The number of rows that should be copied
 * @param[in]   cols                  The number of columns that should be copied
 * @param[in]   allocated_cols        The number of columns that are allocated for each matrix row (may be an overallocation of cols)
 * @param[in]   initial_offset        How much of the host matrix must be skipped at the beginning
 * @param[in]   next_row_offset       How much of the host matrix must be skipped to transition to the next row
 * @param[in]   stream                The stream on which to work
 */
void cp_matrix_from_CUDA(float* matrix, float* dst, int rows, int cols, int allocated_cols, int initial_offset, int next_row_offset, hipStream_t stream)
{
    dst += initial_offset;
    for (int i = 0; i < rows; i++)
    {
        hipMemcpyAsync(dst, matrix, cols * sizeof(float), hipMemcpyDeviceToHost, stream);

        dst += next_row_offset;
        matrix += allocated_cols;
    }
}

/**
 * @brief Given a matrix in global memory, prints it (by copying it to host memory at first)
 *
 * @param[in]  mat              The matrix to print
 * @param[in]  rows             Its number of rows
 * @param[in]  cols             Its number of columns
 * @param[in]  name             The name that should be printed
 * @param[in]  flags            Flags related to highlighting (as per matrix::print)
 * @param[in]  highlight_xs     Flags related to highlighting (as per matrix::print)
 * @param[in]  highlight_ys     Flags related to highlighting (as per matrix::print)
 * @param[in]  highlight_count  Flags related to highlighting (as per matrix::print)
 *
 */
void print_CUDA_matrix(float* mat, int rows, int cols, const char* name, int flags, int* highlight_xs, int* highlight_ys, int highlight_count)
{
    float* mat_host = matrix::alloc(rows, cols, false);
    hipMemcpy(mat_host, mat, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
    matrix::print(mat_host, rows, cols, name, flags, highlight_xs, highlight_ys, highlight_count);
    free(mat_host);
    CUDA_CHECK
}

void copy_matrix_compute_checksum(float* h_mat, float* d_mat, int blockRow, int num_split_row, int blockCol, int num_split_col, int totRows, int totCols, int max_block_rows, int max_block_cols, hipStream_t stream, char name, bool without_checksum)
{
    // copy to device
    int extra = without_checksum ? 0 : 1;
    int size = name == 'A' ? ((max_block_rows + extra) * max_block_cols * sizeof(float)) : (max_block_rows * (max_block_cols + extra) * sizeof(float));
    if (blockCol == num_split_col - 1 || blockRow == num_split_row - 1)
        hipMemsetAsync(d_mat, 0, size, stream);

    int block_rows = CEIL_DIV(totRows, num_split_row);
    int block_cols = CEIL_DIV(totCols, num_split_col);
    int offset = blockRow * max_block_rows * totCols + blockCol * max_block_cols;

    if (blockCol == num_split_col - 1)
        block_cols = totCols - block_cols * blockCol;
    if (blockRow == num_split_row - 1)
        block_rows = totRows - block_rows * blockRow;

    cp_matrix_to_CUDA(h_mat, d_mat, block_rows, block_cols, max_block_cols, offset, totCols, name == 'B' && !without_checksum, stream);

    if (!without_checksum)
    {
        // calculate col checksums for A
        dim3 gridDim = name == 'A' ? dim3(max_block_cols) : dim3(1, max_block_rows);
        dim3 blockDim = name == 'A' ? dim3(1, tileDim.y) : dim3(tileDim.x, 1);
        int sharedMemSize = linearDimToBytes(name == 'A' ? tileDim.y : tileDim.x);
        kernels::compute_checksums<<<gridDim, blockDim, sharedMemSize, stream>>>(d_mat, max_block_rows, max_block_cols, name == 'A' ? ReductionDirection::ALONG_COL : ReductionDirection::ALONG_ROW);
    }

    // Print mat (with checksums)
    if (globals::debugPrint)
        print_CUDA_matrix(
            d_mat,
            max_block_rows + (name == 'A' ? extra : 0),
            max_block_cols + (name == 'A' ? 0 : extra),
            without_checksum ? (name == 'A' ? "A" : "B") : (name == 'A' ? "A (w/ column checksum)" : "B (w/ column checksum)"),
            without_checksum ? 0 : (name == 'A' ? HIGHLIGHT_LAST_ROW : HIGHLIGHT_LAST_COL),
            NULL,
            NULL,
            0);
}

void C_compute_checksum(float* C, ReductionDirection direction, int max_block_cols_B, int max_block_rows_A, hipStream_t stream, float* result_array)
{
    // compute col control checksum

    if (direction == ReductionDirection::ALONG_COL)
    {
        dim3 gridDim = dim3(MAX_BLOCK_COLS_C + 1);
        dim3 blockDim = dim3(1, tileDim.y);
        int sharedMemSize = linearDimToBytes(tileDim.y);
        kernels::compute_checksums<<<gridDim, blockDim, sharedMemSize, stream>>>(C, MAX_BLOCK_ROWS_C, (MAX_BLOCK_COLS_C + 1), ReductionDirection::ALONG_COL, result_array);
    }

    // compute row control checksum
    else
    {
        dim3 gridDim = dim3(1, MAX_BLOCK_ROWS_C + 1);
        dim3 blockDim = dim3(tileDim.x, 1);
        int sharedMemSize = linearDimToBytes(tileDim.x);
        kernels::compute_checksums<<<gridDim, blockDim, sharedMemSize, stream>>>(C, (MAX_BLOCK_ROWS_C + 1), MAX_BLOCK_COLS_C, ReductionDirection::ALONG_ROW, result_array);
    }
}

namespace cuda
{
    void C_mult_check_correct(float* A, float* B, float* C, int rows_A, int cols_B, int* block_rows_C_cur, int* block_cols_C_cur, int C_row, int C_col, int block, int max_block_rows_A, int max_block_cols_A, int max_block_cols_B, hipStream_t stream, hipStream_t streamBis, int num_split_common_dim, int num_split_other_dim, int errors_count, int* error_xs, int* error_ys, float* error_values, bool* result_correct, bool* result_corrected, bool without_error_check)
    {
        int extra = without_error_check ? 0 : 1;

        // rows, cols for dC_cur
        (*block_rows_C_cur) = CEIL_DIV(ROWS_C, num_split_other_dim);
        (*block_cols_C_cur) = CEIL_DIV(COLS_C, num_split_other_dim);

        if (C_row == num_split_other_dim - 1)
            (*block_rows_C_cur) = ROWS_C - (*block_rows_C_cur) * C_row;
        if (C_col == num_split_other_dim - 1)
            (*block_cols_C_cur) = COLS_C - (*block_cols_C_cur) * C_col;

        // compute the actual matrix multiplication as usual

        {
            ScopedTimer timer("tiled matmul", POST);

            dim3 gridDim = dim3(CEIL_DIV(MAX_BLOCK_COLS_C + extra, tileDim.x), CEIL_DIV(MAX_BLOCK_ROWS_C + extra, tileDim.y));
            int sharedMemSize = 2 * dim2ToBytes(tileDim);
            kernels::tiled_matmul<<<gridDim, tileDim, sharedMemSize, stream>>>(A, B, C, max_block_rows_A + extra, max_block_cols_A, max_block_cols_B + extra);

            CUDA_CHECK
        }

        if (without_error_check)
        {
            if (globals::debugPrint)
                print_CUDA_matrix(C, MAX_BLOCK_ROWS_C + extra, MAX_BLOCK_COLS_C, "C", 0, NULL, NULL, 0);
            return;
        }

        float *d_cc_control, *d_rc_control;
        hipMalloc(&d_cc_control, (MAX_BLOCK_COLS_C + 1) * sizeof(float));
        hipMalloc(&d_rc_control, (MAX_BLOCK_ROWS_C + 1) * sizeof(float));

        hipEvent_t C_err_added;
        hipEventCreate(&C_err_added);

        // introduce errors in dC_cur
        {
            ScopedTimer timer("introduce error(s)", POST);

            for (int i = 0; i < errors_count; i++)
            {
                float tmp;
                hipMemcpy(&tmp, C + error_ys[i] * (MAX_BLOCK_COLS_C + 1) + error_xs[i], sizeof(float), hipMemcpyDeviceToHost);
                tmp += error_values[i];
                hipMemcpy(C + error_ys[i] * (MAX_BLOCK_COLS_C + 1) + error_xs[i], &tmp, sizeof(float), hipMemcpyHostToDevice);
            }
            hipEventRecord(C_err_added, stream);

            CUDA_CHECK
        }

        // print dC_cur (with mul checksums)
        if (globals::debugPrint)
            print_CUDA_matrix(C, MAX_BLOCK_ROWS_C + 1, MAX_BLOCK_COLS_C + 1, "C (w/ column checksum)", HIGHLIGHT_LAST_ROW_AND_COL, error_xs, error_ys, errors_count);

        // compute control checksums after mul
        {
            ScopedTimer timer("compute control checksums", POST);

            // compute col control checksum
            C_compute_checksum(C, ReductionDirection::ALONG_COL, max_block_cols_B, max_block_rows_A, stream, d_cc_control);

            // compute row control checksum
            hipStreamWaitEvent(streamBis, C_err_added);
            C_compute_checksum(C, ReductionDirection::ALONG_ROW, max_block_cols_B, max_block_rows_A, streamBis, d_rc_control);

            CUDA_CHECK
        }

        // print control checksums
        if (globals::debugPrint)
        {
            std::vector<int> zeros(errors_count, 0);
            print_CUDA_matrix(d_rc_control, MAX_BLOCK_ROWS_C + 1, 1, "C control row checksum", HIGHLIGHT_LAST_COL, zeros.data(), error_ys, errors_count);
            print_CUDA_matrix(d_cc_control, 1, MAX_BLOCK_COLS_C + 1, "C control column checksum", HIGHLIGHT_LAST_ROW, error_xs, zeros.data(), errors_count);
        }

        // edc

        {
            ScopedTimer timer("error detection (+ correction)", POST);

            bool recompute_vertical_checksums = false;
            bool recompute_horizontal_checksums = false;

            EDCResult edc_res = errors_detect_correct(C, MAX_BLOCK_ROWS_C, MAX_BLOCK_COLS_C, d_cc_control, d_rc_control, stream, streamBis, &recompute_vertical_checksums, &recompute_horizontal_checksums);

            // choice: don't send back the result if it's wrong
            // NOTE: now the result may be partial, since an error will stop the rest
            switch (edc_res)
            {
                case UNCORRECTABLE_ERROR:
                    *result_correct = false;
                    break;

                case CORRECTED_ERROR:
                    *result_corrected = true;
                    break;

                case NO_ERROR:
                    break;
            }

            if (recompute_horizontal_checksums)
                C_compute_checksum(C, ReductionDirection::ALONG_ROW, max_block_cols_B, max_block_rows_A, stream, NULL);

            if (recompute_vertical_checksums)
                C_compute_checksum(C, ReductionDirection::ALONG_COL, max_block_cols_B, max_block_rows_A, stream, NULL);
        }

        hipFree(d_cc_control);
        hipFree(d_rc_control);
        hipEventDestroy(C_err_added);
    }

    EDCResult matmul_ec(float* A, float* B, float* C, int rows_A, int cols_A, int cols_B, int errors_count, int** error_xs, int** error_ys, float** error_values, Strategy strategy, bool without_error_check)
    {
        // How to split the matrices into blocks
        int num_split_common_dim;
        int num_split_other_dim;

        matrix::choose_division(rows_A, cols_A, cols_B, &num_split_common_dim, &num_split_other_dim, strategy);

        // Final sizes of matrices (excluding the checksums)
        int max_block_rows_A = CEIL_DIV(rows_A, num_split_other_dim);
        int max_block_cols_A = CEIL_DIV(cols_A, num_split_common_dim);
        int max_block_cols_B = CEIL_DIV(cols_B, num_split_other_dim);

        // register host pointers as pinned

        hipHostRegister(A, SIZE_A_BYTES, hipHostRegisterDefault);
        hipHostRegister(B, SIZE_B_BYTES, hipHostRegisterDefault);
        hipHostRegister(C, SIZE_C_BYTES, hipHostRegisterDefault);

        CUDA_CHECK

        // allocate device matrices with extra space for checksums A(m+1xn)B(nxp+1) = C(m+1xp+1)

        int extra = without_error_check ? 0 : 1;
        int size_A_ec = (max_block_rows_A + extra) * max_block_cols_A * sizeof(float);
        int size_B_ec = MAX_BLOCK_ROWS_B * (max_block_cols_B + extra) * sizeof(float);
        int size_C_ec = (MAX_BLOCK_ROWS_C + extra) * (MAX_BLOCK_COLS_C + extra) * sizeof(float);

        float *dA1, *dB1, *dC1, *dA2, *dB2, *dC2;

        switch (strategy)
        {
            case preloadAB_deferUnloadC:
            case parallelMul:
                hipMalloc(&dC2, size_C_ec);

            case preloadAB:
                if (strategy != parallelMul)
                    hipMalloc(&dA2, size_A_ec);
                hipMalloc(&dB2, size_B_ec);

            case simple:
                hipMalloc(&dA1, size_A_ec);
                hipMalloc(&dB1, size_B_ec);
                hipMalloc(&dC1, size_C_ec);
        }

        float* dA_cur = dA1;
        float* dA_alt = dA2;

        float* dB_cur = dB1;
        float* dB_alt = dB2;

        float* dC_cur = dC1;
        float* dC_alt = dC2;

        CUDA_CHECK

        // create streams for parallel executions
        hipStream_t stream_A1;
        hipStream_t stream_B1;
        hipStream_t stream_C1;
        hipStream_t stream_C1bis;
        hipStream_t stream_A2;
        hipStream_t stream_B2;
        hipStream_t stream_C2;
        hipStream_t stream_C2bis;

        hipStream_t* stream_A_cur = &stream_A1;
        hipStream_t* stream_A_alt = &stream_A2;

        hipStream_t* stream_B_cur = &stream_B1;
        hipStream_t* stream_B_alt = &stream_B2;

        hipStream_t* stream_C_cur = &stream_C1;
        hipStream_t* stream_C_alt = &stream_C2;

        hipStream_t* stream_Cbis_cur = &stream_C1bis;
        hipStream_t* stream_Cbis_alt = &stream_C2bis;

        switch (strategy)
        {
            case preloadAB_deferUnloadC:
            case parallelMul:
                hipStreamCreate(&stream_C2);
                hipStreamCreate(&stream_C2bis);

            case preloadAB:
                hipStreamCreate(&stream_A2);
                hipStreamCreate(&stream_B2);

            case simple:
                hipStreamCreate(&stream_A1);
                hipStreamCreate(&stream_B1);
                hipStreamCreate(&stream_C1);
                hipStreamCreate(&stream_C1bis);
        }

        // result
        bool result_correct = true;
        bool result_corrected = false;
        bool result_correct_alt = true;
        bool result_corrected_alt = false;

        //
        int offset;

        int block_rows_C_cur;
        int block_cols_C_cur;

        int block_rows_C_alt;
        int block_cols_C_alt;

        // Sync events
        hipEvent_t A_copied;
        hipEvent_t B_copied;
        hipEvent_t A_alt_copied;
        hipEvent_t B_alt_copied;

        hipEvent_t A_can_be_overwritten;
        hipEvent_t B_can_be_overwritten;
        hipEvent_t B_alt_can_be_overwritten;

        CUDA_CREATE_RECORD_EVENT(A_can_be_overwritten, *stream_A_cur);
        CUDA_CREATE_RECORD_EVENT(B_can_be_overwritten, *stream_B_cur);
        if (strategy == parallelMul)
            CUDA_CREATE_RECORD_EVENT(B_alt_can_be_overwritten, *stream_B_alt);

        if (strategy != simple && strategy != parallelMul)
        {
            copy_matrix_compute_checksum(A, dA_cur, 0, num_split_other_dim, 0, num_split_common_dim, rows_A, cols_A, max_block_rows_A, max_block_cols_A, *stream_A_cur, 'A', without_error_check);
            CUDA_CREATE_RECORD_EVENT(A_copied, *stream_A_cur);

            copy_matrix_compute_checksum(B, dB_cur, 0, num_split_common_dim, 0, num_split_other_dim, ROWS_B, cols_B, MAX_BLOCK_ROWS_B, max_block_cols_B, *stream_B_cur, 'B', without_error_check);
            CUDA_CREATE_RECORD_EVENT(B_copied, *stream_B_cur);
        }

        for (int C_row = 0; C_row < num_split_other_dim && result_correct; C_row++)
        {
            for (int C_col = 0; C_col < num_split_other_dim && result_correct; C_col += (strategy == parallelMul ? 2 : 1))
            {
                hipMemsetAsync(dC_cur, 0, size_C_ec, *stream_C_cur);
                if (strategy == parallelMul && C_col + 1 < num_split_other_dim)
                    hipMemsetAsync(dC_alt, 0, size_C_ec, *stream_C_alt);

                for (int block = 0; block < num_split_common_dim && result_correct; block++)
                {

                    // send matrices to device and calculate checksums in parallel on different streams

                    // stream1: copyA, checkA
                    // stream2: copyB, checkB
                    // copies are sent to possibly the same copy queue, kernels to the same kernel queue (especially if only one queue per category exists)
                    // we use depth-first issue order: copyA - checkA - copyB - checkB
                    // breadth-first issue order would be: copyA - copyB - checkA - checkB
                    // we found that depth-first gives better performance

                    {
                        ScopedTimer timer("A,B to device + compute input checksums", POST);
                        if (strategy == simple)
                        {
                            CUDA_WAIT_DESTROY_EVENT(A_can_be_overwritten, *stream_A_cur, true)
                            copy_matrix_compute_checksum(A, dA_cur, C_row, num_split_other_dim, block, num_split_common_dim, rows_A, cols_A, max_block_rows_A, max_block_cols_A, *stream_A_cur, 'A', without_error_check);
                            CUDA_CREATE_RECORD_EVENT(A_copied, *stream_A_cur);

                            CUDA_WAIT_DESTROY_EVENT(B_can_be_overwritten, *stream_B_cur, true)
                            copy_matrix_compute_checksum(B, dB_cur, block, num_split_common_dim, C_col, num_split_other_dim, ROWS_B, cols_B, MAX_BLOCK_ROWS_B, max_block_cols_B, *stream_B_cur, 'B', without_error_check);
                            CUDA_CREATE_RECORD_EVENT(B_copied, *stream_B_cur);
                        }
                        else if (strategy == parallelMul)
                        {
                            CUDA_WAIT_DESTROY_EVENT(A_can_be_overwritten, *stream_A_cur, true)
                            copy_matrix_compute_checksum(A, dA_cur, C_row, num_split_other_dim, block, num_split_common_dim, rows_A, cols_A, max_block_rows_A, max_block_cols_A, *stream_A_cur, 'A', without_error_check);
                            CUDA_CREATE_RECORD_EVENT(A_copied, *stream_A_cur);

                            CUDA_WAIT_DESTROY_EVENT(B_can_be_overwritten, *stream_B_cur, true)
                            copy_matrix_compute_checksum(B, dB_cur, block, num_split_common_dim, C_col, num_split_other_dim, ROWS_B, cols_B, MAX_BLOCK_ROWS_B, max_block_cols_B, *stream_B_cur, 'B', without_error_check);
                            CUDA_CREATE_RECORD_EVENT(B_copied, *stream_B_cur);

                            if (C_col + 1 < num_split_other_dim)
                            {
                                CUDA_WAIT_DESTROY_EVENT(B_alt_can_be_overwritten, *stream_B_alt, true)
                                copy_matrix_compute_checksum(B, dB_alt, block, num_split_common_dim, C_col + 1, num_split_other_dim, ROWS_B, cols_B, MAX_BLOCK_ROWS_B, max_block_cols_B, *stream_B_alt, 'B', without_error_check);
                                CUDA_CREATE_RECORD_EVENT(B_alt_copied, *stream_B_alt);
                            }
                        }
                        else
                        {
                            // if strategy pre-loads A and B, and this is not the last iteration, pre-load the next A, B
                            if (block != (num_split_common_dim - 1) || C_row != (num_split_other_dim - 1) || C_col != (num_split_other_dim - 1))
                            {
                                int next_block = block + 1;
                                int next_C_col = C_col;
                                int next_C_row = C_row;
                                if (next_block == num_split_common_dim)
                                {
                                    next_block = 0;
                                    next_C_col = C_col + 1;
                                    if (next_C_col == num_split_other_dim)
                                    {
                                        next_C_col = 0;
                                        next_C_row = C_row + 1;
                                    }
                                }

                                CUDA_WAIT_DESTROY_EVENT(A_can_be_overwritten, *stream_A_alt, true)
                                copy_matrix_compute_checksum(A, dA_alt, next_C_row, num_split_other_dim, next_block, num_split_common_dim, rows_A, cols_A, max_block_rows_A, max_block_cols_A, *stream_A_alt, 'A', without_error_check);
                                CUDA_CREATE_RECORD_EVENT(A_alt_copied, *stream_A_alt);

                                CUDA_WAIT_DESTROY_EVENT(B_can_be_overwritten, *stream_B_alt, true)
                                copy_matrix_compute_checksum(B, dB_alt, next_block, num_split_common_dim, next_C_col, num_split_other_dim, ROWS_B, cols_B, MAX_BLOCK_ROWS_B, max_block_cols_B, *stream_B_alt, 'B', without_error_check);
                                CUDA_CREATE_RECORD_EVENT(B_alt_copied, *stream_B_alt);
                            }
                        }

                        CUDA_CHECK
                    }

                    int error_id = block + C_col * num_split_common_dim + C_row * num_split_common_dim * num_split_other_dim;

                    CUDA_WAIT_DESTROY_EVENT(A_copied, *stream_C_cur, strategy != parallelMul || C_col + 1 >= num_split_other_dim)
                    CUDA_WAIT_DESTROY_EVENT(B_copied, *stream_C_cur, true)

                    C_mult_check_correct(dA_cur, dB_cur, dC_cur, rows_A, cols_B, &block_rows_C_cur, &block_cols_C_cur, C_row, C_col, block, max_block_rows_A, max_block_cols_A, max_block_cols_B, *stream_C_cur, *stream_Cbis_cur, num_split_common_dim, num_split_other_dim, errors_count, error_xs[error_id], error_ys[error_id], error_values[error_id], &result_correct, &result_corrected, without_error_check);

                    if (strategy != parallelMul || C_col + 1 >= num_split_other_dim)
                        CUDA_CREATE_RECORD_EVENT(A_can_be_overwritten, *stream_C_cur);
                    CUDA_CREATE_RECORD_EVENT(B_can_be_overwritten, *stream_C_cur);

                    if (strategy == parallelMul && C_col + 1 < num_split_other_dim)
                    {
                        CUDA_WAIT_DESTROY_EVENT(A_copied, *stream_C_alt, true)
                        CUDA_WAIT_DESTROY_EVENT(B_alt_copied, *stream_C_alt, true)

                        error_id = block + (C_col + 1) * num_split_common_dim + C_row * num_split_common_dim * num_split_other_dim;
                        C_mult_check_correct(dA_cur, dB_alt, dC_alt, rows_A, cols_B, &block_rows_C_alt, &block_cols_C_alt, C_row, C_col + 1, block, max_block_rows_A, max_block_cols_A, max_block_cols_B, *stream_C_alt, *stream_Cbis_alt, num_split_common_dim, num_split_other_dim, errors_count, error_xs[error_id], error_ys[error_id], error_values[error_id], &result_correct_alt, &result_corrected_alt, without_error_check);

                        CUDA_CREATE_RECORD_EVENT(A_can_be_overwritten, *stream_C_cur);
                        CUDA_CREATE_RECORD_EVENT(B_alt_can_be_overwritten, *stream_C_alt);
                    }

                    switch (strategy)
                    {
                        case preloadAB_deferUnloadC:
                        case preloadAB:
                            SWAP(dA_cur, dA_alt)
                            SWAP(dB_cur, dB_alt)
                            SWAP(stream_A_cur, stream_A_alt)
                            SWAP(stream_B_cur, stream_B_alt)
                            SWAP(A_copied, A_alt_copied)
                            SWAP(B_copied, B_alt_copied)
                    }
                }
                // send back result (without checksums)

                {
                    ScopedTimer timer("C to host", POST);

                    offset = C_row * MAX_BLOCK_ROWS_C * COLS_C + C_col * MAX_BLOCK_COLS_C;
                    int offset2 = C_row * MAX_BLOCK_ROWS_C * COLS_C + (C_col + 1) * MAX_BLOCK_COLS_C;

                    switch (strategy)
                    {
                        case preloadAB_deferUnloadC:
                            SWAP(dC_cur, dC_alt)
                            SWAP(stream_C_cur, stream_C_alt)
                            SWAP(stream_Cbis_cur, stream_Cbis_alt)
                            SWAP(block_rows_C_cur, block_rows_C_alt)
                            SWAP(block_cols_C_cur, block_cols_C_alt)

                            cp_matrix_from_CUDA(dC_alt, C, block_rows_C_alt, block_cols_C_alt, MAX_BLOCK_COLS_C + extra, offset, COLS_C, *stream_C_alt);
                            break;

                        case parallelMul:
                            if (C_col + 1 < num_split_other_dim)
                                cp_matrix_from_CUDA(dC_alt, C, block_rows_C_alt, block_cols_C_alt, MAX_BLOCK_COLS_C + extra, offset2, COLS_C, *stream_C_alt);

                        case preloadAB:
                        case simple:
                            cp_matrix_from_CUDA(dC_cur, C, block_rows_C_cur, block_cols_C_cur, MAX_BLOCK_COLS_C + extra, offset, COLS_C, *stream_C_cur);
                    }

                    CUDA_CHECK
                }
            }
        }

        // cleanup:

        switch (strategy)
        {
            case preloadAB_deferUnloadC:
            case parallelMul:
                hipStreamDestroy(stream_C2);
                hipStreamDestroy(stream_C2bis);

            case preloadAB:
                hipStreamDestroy(stream_A2);
                hipStreamDestroy(stream_B2);

            case simple:
                hipStreamDestroy(stream_A1);
                hipStreamDestroy(stream_B1);
                hipStreamDestroy(stream_C1);
                hipStreamDestroy(stream_C1bis);
        }

        switch (strategy)
        {
            case preloadAB_deferUnloadC:
            case parallelMul:
                hipFree(dC2);

            case preloadAB:
                if (strategy != parallelMul)
                    hipFree(dA2);
                hipFree(dB2);

            case simple:
                hipFree(dA1);
                hipFree(dB1);
                hipFree(dC1);
        }

        hipHostUnregister(A);
        hipHostUnregister(B);
        hipHostUnregister(C);

        CUDA_CHECK

        if (!result_correct)
            return UNCORRECTABLE_ERROR;
        return result_corrected ? CORRECTED_ERROR : NO_ERROR;
    }
}
