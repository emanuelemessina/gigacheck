#include "cuda.cuh"

namespace cuda
{
    Info getInfo()
    {
        hipDeviceProp_t deviceProp;
        int device;
        hipGetDevice(&device);
        hipGetDeviceProperties(&deviceProp, device);
        Info info{
            device,
            deviceProp.name,
            deviceProp.multiProcessorCount,
            deviceProp.totalGlobalMem,
            deviceProp.sharedMemPerBlock,
            deviceProp.sharedMemPerMultiprocessor};
        return info;
    }
}
